// function definitions for creating space on device for atoms

#include "device_space.cuh"

void make_device_space( Device_Atoms * dev_atoms, Device_Box * dev_box, Box host_box, int natoms )
{

	// make room for atoms on device, the positions will be copied before each timestep
	size_t size = sizeof(float)*natoms;
	hipMalloc( dev_atoms->getDevPtr(1) , size );
	hipMalloc( dev_atoms->getDevPtr(2) , size );
	hipMalloc( dev_atoms->getDevPtr(3) , size );
	hipMalloc( dev_atoms->getDevPtr(4) , size );
	
	// make room for box dimensions on device
	hipMalloc( dev_box->getDevPtr(1) , 3*sizeof(float) );
	hipMalloc( dev_box->getDevPtr(2) , 3*sizeof(float) );
	
	// copy the box info to the device (this only needs to be done once)
	float ** d_box_len = dev_box->getDevPtr(1);
	float ** d_inv_box_len = dev_box->getDevPtr(2);
	float * h_box_len = host_box.getLength(1);
	float * h_box_len_inv = host_box.getLength(2);
	hipMemcpy(*d_box_len,h_box_len,3*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(*d_inv_box_len,h_box_len_inv,3*sizeof(float),hipMemcpyHostToDevice);

}

void clear_device_space( float ** xtmp, float ** ytmp, float ** ztmp, float ** rho, 
                         float ** Box_len, float ** Box_len_inv )
{

    // free up all the GPU memory
	hipFree( *xtmp );
	hipFree( *ytmp );
	hipFree( *ztmp );
	hipFree( *rho );
	hipFree( *Box_len );
	hipFree( *Box_len_inv );

}

void make_space_compute( float ** denergy, float ** dforce, int n )
{

	// make room on the GPU for the energy and force associated with each atom 
	size_t size = sizeof(float)*n;
	hipMalloc( denergy , size ); // total potential energy
	hipMalloc( dforce , size ); // 3d force
	
	//hipError_t result;
	//result = hipMalloc( dforce , 3*sizeof(float) ); // 3d force
	//cout << "comments from hipMalloc: " << hipGetErrorString(result) << endl;


} 

void clear_space_compute( float ** denergy, float ** dforce)
{

	// free space on GPU
	hipFree( *denergy );
	hipFree( *dforce );

}
