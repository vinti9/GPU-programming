#include "hip/hip_runtime.h"
// CUDA functions/kernels for computing the 
// TB-SMA energy/force

#include <iostream>
using std::cout;
using std::endl;

#include "TBSMA_GPU_Compute.cuh"

#define BLOCK_SIZE 64 // this block size is optimized for larger systems

__global__ void TBSMA_kernel_1( float * xpos, float * ypos, float * zpos, float * rho, 
                                           int natoms, float * box_len, float * inv_box_len )
{

	float param_ro = 2.88426;
	float param_2q = 2.0 * 4.036;
	float r_cutsq = 5.8 * 5.8;
	float energy_band_term = 0.0;
	
	volatile int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx >= natoms) return;
	
	// read in particle positions from device global memory
	// textured reads may improve performance here
	float xi = xpos[ idx ];
	float yi = ypos[ idx ];
	float zi = zpos[ idx ];
	float dx,dy,dz,rsq;
	
	// loop over all atoms, not using Newton's 2nd law here 
	// because that requires communication between threads, it's
	// more efficient to just re-calculate the energy between
	// two atoms.
	//
	// A neighborlist would be more efficient here but for 
	// simplicity I am not using one
	//
	for (int i = 0; i < natoms ; i++)
	{
		
		if (idx != i) // don't count self-interaction
		{
		
			dx = xi - xpos[ i ];
			dy = yi - ypos[ i ];
			dz = zi - zpos[ i ];
			
			// periodic boundary conditions
			dx -= box_len[0] * rintf( dx * inv_box_len[0] );
			dy -= box_len[1] * rintf( dy * inv_box_len[1] );
			dz -= box_len[2] * rintf( dz * inv_box_len[2] );
		
			rsq = dx*dx + dy*dy + dz*dz;
			
			if (rsq < r_cutsq)
			{
			
				float r = sqrt( rsq );
				float r_modified = 1.0 - r / param_ro;
				energy_band_term += exp( param_2q * r_modified );
			
			}
			
		
		}
	
	}
	
	rho[ idx ] = energy_band_term;

}

__global__ void TBSMA_kernel_2( float * xpos, float * ypos, float * zpos, float * rho, 
                                           int natoms, float * box_len, float * inv_box_len,
										   float * energy_device, float * force_device )
{

	float param_P = 10.229;
	float param_q = 4.036;
	float param_ro = 2.88426;
	float param_2q = 2.0 * 4.036;
	float r_cutsq = 5.8 * 5.8;
	
	float param_A = 0.2061;
	float param_B = 1.790;
	
	float param_BQinvro = param_B * param_q / param_ro;
	float param_APinvro = 2.0 * param_A * param_P / param_ro;
	
	volatile int idx = blockIdx.x * blockDim.x + threadIdx.x;
	energy_device[ idx ] = 0.0;
	force_device[ idx ] = 0.0;
	
	if (idx >= natoms) return;
	
	// read in particle positions from device global memory
	// textured reads may improve performance here
	float xi = xpos[ idx ];
	float yi = ypos[ idx ];
	float zi = zpos[ idx ];
	float dx,dy,dz,rsq;
	float fxi = 0.0;
	float fyi = 0.0;
	float fzi = 0.0;
	float m_pe = 0.0;
	float pairForce = 0.0;
	float pair_eng = 0.0;
	
	// rho from previous kernel
	float rho_i = rho[ idx ];
	float sqrt_rho_i = sqrtf( rho_i );
	float inv_sqrt_rho_i = 1.0 / sqrt_rho_i;
	
	// loop over all atoms
	for (int i = 0; i < natoms ; i++)
	{
		
		if (idx != i) // don't cout self-interaction
		{
		
			dx = xi - xpos[ i ];
			dy = yi - ypos[ i ];
			dz = zi - zpos[ i ];
			
			// periodic boundary conditions
			dx -= box_len[0] * rint( dx * inv_box_len[0] );
			dy -= box_len[1] * rint( dy * inv_box_len[1] );
			dz -= box_len[2] * rint( dz * inv_box_len[2] );
		
			rsq = dx*dx + dy*dy + dz*dz;
			
			if (rsq < r_cutsq)
			{
			
				float r = sqrt( rsq );
				float rinv = 1.0 / r;
				float r_modified = 1.0 - r / param_ro;
				
				float energy_repulsive = exp( param_P * r_modified );
				float energy_temp = exp( param_2q * r_modified );
				
				float rho_j = rho[ i ];
				float inv_sqrt_rho_j = 1.0 / sqrt( rho_j );
				
				pair_eng = energy_repulsive * param_A; // repulsive, pairwise energy
				pairForce = -param_BQinvro * energy_temp * (inv_sqrt_rho_i + inv_sqrt_rho_j) + energy_repulsive * param_APinvro;
                pairForce *= rinv;
				
				fxi += dx * pairForce ;
				fyi += dy * pairForce ;
				fzi += dz * pairForce ;
				m_pe += pair_eng;

			
			}
			
		
		}
	
	}
	
	energy_device[ idx ] = m_pe - param_B * sqrt_rho_i; // PE of this thread's atom
	force_device[ idx ] = sqrt( fxi*fxi + fyi*fyi + fzi*fzi ); // Force acting on this thread's atom

}

__global__ void Reduce_Energy( float * energy, float * force )
{

	__shared__ float Es[BLOCK_SIZE];
	__shared__ float Fs[BLOCK_SIZE];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	// load global data into shared memory
	Es[ tid ] = energy[ i ];
	Fs[ tid ] = force[ i ];
	
	__syncthreads(); // wait for everything to load
	
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
		
			Es[ tid ] += Es[ tid + s ];
			Fs[ tid ] += Fs[ tid + s ];
		
		}
		__syncthreads();  // wait for all threads to finish
	
	}
	
	// reduce in shared memory (naive version)
	//for (unsigned int s=1; s < blockDim.x; s *= 2) {
	
	//	if ( tid % (2*s) == 0 ) {
	//		Es[ tid ] += Es[ tid + s ];
	//		Fs[ tid ] += Fs[ tid + s ];
	//	}
	//	__syncthreads(); 
	
	//}
	
	// copy results back to the energy and force arrays,
	// I'll sum up the results from each block on the CPU
	energy[ i ] = Es[ tid ];
	force[ i ] = Fs[ tid ];
	

}

// this function calls the CUDA kernels that calculate the TB-SMA energy/force
void gpu_compute_TBSMA( Atoms * host_atoms , Device_Atoms * device_atoms, Device_Box * device_box, float * host_energy, 
                        float * host_force, float ** device_energy, float ** device_force, bool print_step )
{

    //
	// (1) copy atom positions to device
	// (2) compute the energy/force on the device
	// (3) copy the energy/force back to the CPU
	//
	
	
	float * h_xtmp = host_atoms->getCoords(1);
	float * h_ytmp = host_atoms->getCoords(2);
	float * h_ztmp = host_atoms->getCoords(3);
	float ** d_xtmp = device_atoms->getDevPtr(1);
	float ** d_ytmp = device_atoms->getDevPtr(2);
	float ** d_ztmp = device_atoms->getDevPtr(3);
	float ** rho = device_atoms->getDevPtr(4);
	float ** d_box_len = device_box->getDevPtr(1);
	float ** d_inv_box_len = device_box->getDevPtr(2);
	
	int n = host_atoms->printNatoms();
	
	size_t size = sizeof(float)*n;
	
	// copy atom coordinates to the GPU every time step
	//hipError_t result;
	//result = hipMemcpy(*d_xtmp,h_xtmp,size,hipMemcpyHostToDevice);
	hipMemcpy(*d_xtmp,h_xtmp,size,hipMemcpyHostToDevice);
	hipMemcpy(*d_ytmp,h_ytmp,size,hipMemcpyHostToDevice);
	hipMemcpy(*d_ztmp,h_ztmp,size,hipMemcpyHostToDevice);
	
	int blocks = (int)ceil((float)n / (float)BLOCK_SIZE);

    dim3 grid(blocks, 1, 1);
    dim3 threads(BLOCK_SIZE, 1, 1);
	
	// call the first kernel for calculating the electron density
	TBSMA_kernel_1<<< grid, threads>>>( *d_xtmp, *d_ytmp, *d_ztmp, *rho, n, *d_box_len, *d_inv_box_len );
	
	// call the second kernel for calculating the energy/force
	TBSMA_kernel_2<<< grid, threads>>>( *d_xtmp, *d_ytmp, *d_ztmp, *rho, n, *d_box_len, *d_inv_box_len,
	                                    *device_energy, *device_force );
										
	
	// if energy/force need to be output this timestep,
	// copy resulting energy/force to the CPU
	if (print_step) {
	
		// get the total energy/force by summing up the energy/force
		// acting on individual atoms									
		Reduce_Energy<<< grid, threads>>>( *device_energy, *device_force );
		
		// Copy results to CPU
		hipMemcpy(host_energy,*device_energy,size,hipMemcpyDeviceToHost);
		hipMemcpy(host_force,*device_force,size,hipMemcpyDeviceToHost);
		
		for (int j=1; j<blocks; j++) { // add up the block sums
			host_energy[0] += host_energy[ j*BLOCK_SIZE ];
			host_force[0] += host_force[ j*BLOCK_SIZE ];
		}
	
	}

}
